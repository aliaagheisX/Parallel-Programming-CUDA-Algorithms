#include "hip/hip_runtime.h"
#include <unistd.h>
#include<iostream>
#include<stdint-gcc.h>
#include<error.h>
#include<algorithm>

// 1. radix sort [x]
// 2. arg radix sort [x]
// 3. ASC, DESC [x]
// 4. float, signed numbers [x]
// 5. transpose SUM to allocate for every batch -> compine
// 6. streams ---> kill me please

#define CUDA_CHECK(call) { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        fprintf(stderr, "CUDA error in %s at line %d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
        exit(EXIT_FAILURE); \
    } \
}

#define CUDA_CHECK_LAST() { \
    hipError_t err = hipGetLastError(); \
    if (err != hipSuccess) { \
        fprintf(stderr, "CUDA error in %s at line %d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
        exit(EXIT_FAILURE); \
    } \
}


#define BLOCK_DIM 4
// #define MAX_BATCH_SZ 32764
#define COARSING_FACTOR 2
#define RADIX 2
#define BUCKET_SZ 4 // 1<<RADIX
// get ith bit in x
#define GET_BIT(x,i) ((x >> i) & 1LL)
// get mask 000111 if radix = 3
#define MASK_ZERO ((1 << (RADIX)) - 1)
// get mask of iter 000111000 if iter = 1
#define MASK_ITER(iter) (MASK_ZERO << (iter*RADIX))
// get radix for certain iter
#define GET_RADIX_KEY(x,iter) ((x>>(iter*RADIX)) & MASK_ZERO)

#define CEIL_DIV(X, Y) ((X + Y - 1)/(Y))
#define CEIL_DIVI(X, Y) int((X + Y - 1)/(Y))

#define NUM_BANKS 32
#define LOG_NUM_BANKS 5
#define CONFLICT_FREE_OFFSET(n)((n) >> NUM_BANKS + (n) >> (2 * LOG_NUM_BANKS))

#define GET_IDX(y, x, cols) (x + y*cols)


__global__  void int_to_uint32(int* arr, uint32_t* res, int N) {
    // Flip the sign bit to make negative numbers come before positive ones in unsigned space
    int global_idx = threadIdx.x + blockDim.x * blockIdx.x;
    if(global_idx < N) {
        res[global_idx] = static_cast<uint32_t>(arr[global_idx]) ^ 0x80000000;
    }
}

__global__  void uint32_to_int(uint32_t* arr, int* res, int N) {
    // reverse process
    int global_idx = threadIdx.x + blockDim.x * blockIdx.x;
    if(global_idx < N) {
        res[global_idx] = static_cast<int32_t>(arr[global_idx] ^ 0x80000000);
    }
}

__global__  void float_to_uint32(float* arr, uint32_t* res, int N) {
    // reverse process
    int global_idx = threadIdx.x + blockDim.x * blockIdx.x;
    if(global_idx < N) {
        uint32_t bits = *reinterpret_cast<uint32_t*>(&arr[global_idx]);
        res[global_idx] = (bits & 0x80000000) ? ~bits : (bits ^ 0x80000000);
    }
}

__global__  void uint32_to_float(uint32_t* arr, float* res, int N) {
    // reverse process
    int global_idx = threadIdx.x + blockDim.x * blockIdx.x;
    if(global_idx < N) {
        uint32_t x = arr[global_idx];
        uint32_t bits = (x & 0x80000000) ? (x ^ 0x80000000) : ~x;
        res[global_idx] = *reinterpret_cast<float*>(&bits);
    }
}

struct SortOp
{
    __device__ virtual inline int get_radix(int x, int iter) const = 0;
    __device__ virtual inline int get_bit(int x, int bit) const = 0;
};
struct AscOp : SortOp
{
    __device__ inline int get_radix(int x, int iter) const override { return GET_RADIX_KEY(x, iter);  }
    __device__ inline int get_bit(int x, int bit) const override { return GET_BIT(x, bit); };
};
struct DescOp : SortOp
{
    __device__ inline int get_radix(int x, int iter) const override { return MASK_ZERO - GET_RADIX_KEY(x, iter);  }
    __device__ inline int get_bit(int x, int bit) const override { return 1-GET_BIT(x, bit); };
};

__device__ void add_prev_block_bank(uint32_t* res, uint32_t* sh_mem, uint32_t actualBlockIdx, const uint32_t sz, const uint32_t arr_idx, uint32_t *blocks_finished) {
    // handle block 0
    __syncthreads();
    // wait for previous
    __shared__ int prev_sum;
    while(atomicAdd(blocks_finished, 0) != actualBlockIdx) {}
    if(threadIdx.x == 0) {
        prev_sum = actualBlockIdx == 0 ? 0 : res[2*blockDim.x - 1 + 2 * blockDim.x * (actualBlockIdx - 1)];
    }
    __syncthreads();
    // let's go & sum previous
    // 3. write output
    int ai = threadIdx.x; ai += CONFLICT_FREE_OFFSET(ai);
    int bi = threadIdx.x + blockDim.x; bi += CONFLICT_FREE_OFFSET(bi);
    // write once in result
    if(arr_idx < sz)
        res[arr_idx] = sh_mem[ai] + prev_sum;
    if(arr_idx + blockDim.x < sz)
        res[arr_idx + blockDim.x] = sh_mem[bi] + prev_sum;

    __syncthreads();
    if(threadIdx.x == blockDim.x - 1) {
        __threadfence();
        atomicAdd(blocks_finished, 1);
    }    
}


__global__ void balanced_bank_kernel(uint32_t* arr, uint32_t* res, const uint32_t sz, uint32_t* block_counter, uint32_t *blocks_finished) {
    // 1. get actual blockIdx
    __shared__ int actualBlockIdx;
    if(threadIdx.x == 0) {
        actualBlockIdx = atomicAdd(block_counter, 1);
    }
    __syncthreads(); // ensure all threads see updated



    const uint32_t arr_idx = threadIdx.x + 2 * blockDim.x * actualBlockIdx; // blockDim.x - 1 + 2 * blockDim.x * actualBlockIdx
    const uint32_t th = threadIdx.x;
    __shared__ uint32_t sh_mem[2 * BLOCK_DIM + 2 * BLOCK_DIM / NUM_BANKS]; // add padding to sh_mem
    // 1. get data into sh_mem
    int ai = th; ai += CONFLICT_FREE_OFFSET(ai);
    int bi = th + blockDim.x; bi += CONFLICT_FREE_OFFSET(bi);
    sh_mem[ai] = arr_idx >= sz ? 0 : arr[arr_idx]; // [arr[0], arr[1], arr[2], arr[3]]
    sh_mem[bi] = arr_idx  + blockDim.x >= sz ? 0 : arr[arr_idx  + blockDim.x];
    __syncthreads();
    // 2. upsweep
    uint64_t offset = 1;
    for(uint32_t d = blockDim.x; d > 0; d>>= 1) {
        ai = 2*offset*(th + 1) - 1; ai += CONFLICT_FREE_OFFSET(ai);
        bi = 2*offset*(th + 1) - offset - 1; bi += CONFLICT_FREE_OFFSET(bi);
        if(th < d) {
            sh_mem[ai] += sh_mem[bi];
        }
        offset <<= 1;
        __syncthreads();
    }

    // 3. downsweep
    for(uint32_t d = 1; d <= blockDim.x; d<<= 1) {
        offset >>= 1;
        ai = 2*offset*(th + 1) + offset - 1;  ai += CONFLICT_FREE_OFFSET(ai);
        bi = 2*offset*(th + 1) - 1;  bi += CONFLICT_FREE_OFFSET(bi);
        if(ai < 2 * BLOCK_DIM + 2 * BLOCK_DIM / NUM_BANKS)  {
            sh_mem[ai] += sh_mem[bi];
        }
        __syncthreads();
    }
    
    add_prev_block_bank(res, sh_mem, actualBlockIdx, sz, arr_idx, blocks_finished);
}


// __device__ int COUNTER[CEIL_DIV(MAX_BATCH_SZ, BLOCK_DIM) * BUCKET_SZ];//should be accessed [BLOCKIdx * BACKET_SZ + radix]

__device__ bool scan_inefficent(uint32_t sh_mem[][BLOCK_DIM+1]) {
    bool buff_idx = 0; // buffer
    const uint32_t thidx = threadIdx.x + 1;
    for(uint32_t offset = 1; offset <= BLOCK_DIM; offset <<= 1) {
        if(thidx >= offset) {
            sh_mem[!buff_idx][thidx] = sh_mem[buff_idx][thidx] + sh_mem[buff_idx][thidx - offset];
        } else {
            sh_mem[!buff_idx][thidx] = sh_mem[buff_idx][thidx];
        }
        buff_idx = !buff_idx;
        __syncthreads();
    }
    return buff_idx;
}

template <typename OP>
__device__ void one_bit_sort(uint32_t* sh_A, uint32_t* res, uint32_t* old_idxs, uint32_t *new_idxs, uint32_t sh_mem[][BLOCK_DIM+1], const uint32_t bit, OP& op) {
    // 1. count
    sh_mem[0][0] = 0;
    sh_mem[0][threadIdx.x+1] = op.get_bit(sh_A[threadIdx.x], bit);
    __syncthreads();
    // 2. scan
    bool buff_idx = scan_inefficent(sh_mem);
    // 3. gather
    // for zeros it's my index - num of ones left me
    // for one it's (total_size - one in total + ones on left) 
    int num_ones = sh_mem[buff_idx][BLOCK_DIM];
    int ones_left = sh_mem[buff_idx][threadIdx.x];

    int new_idx = op.get_bit(sh_A[threadIdx.x], bit) ? 
                  (BLOCK_DIM - num_ones + ones_left) : // For 1s: place after all 0s
                  (threadIdx.x - ones_left);
    res[ new_idx ] = sh_A[threadIdx.x];
    new_idxs[ new_idx ] = old_idxs[threadIdx.x];
}
// global_counter[radix][blockIdx.x]
__device__ void update_glob_buckets(uint32_t* A, uint32_t* local_counter, const uint32_t iter, const uint32_t N, const uint32_t global_idx, uint32_t* global_counter, SortOp& op) {
    if(global_idx < N) {
        int radix = op.get_radix(A[threadIdx.x], iter);
        atomicAdd(&local_counter[radix], 1);
    }
    __syncthreads();
    if(threadIdx.x < BUCKET_SZ) {
        global_counter[GET_IDX(threadIdx.x, blockIdx.x, gridDim.x)] = local_counter[threadIdx.x];
    }
}

template <typename OP>
__global__ void radix_sort_local_kerenl(uint32_t *A, uint32_t *res, uint32_t* old_idxs, uint32_t *new_idxs, uint32_t* global_counter, const uint32_t N, const uint32_t iter) {
    OP op;
    __shared__ uint32_t sh_mem[2][BLOCK_DIM+1]; // 2 blocks swaped every time (+1 since it's exclusive sum)
    __shared__ uint32_t sh_A[2][BLOCK_DIM];
    __shared__ uint32_t sh_idxs[2][BLOCK_DIM];
    __shared__ uint32_t local_counter[BUCKET_SZ];
    // 1. get data into local memory
    if(threadIdx.x == 0) {
        sh_mem[0][0] = 0;
        sh_mem[1][0] = 0;
    }
    if(threadIdx.x < BUCKET_SZ) local_counter[threadIdx.x] = 0;
    int global_idx = threadIdx.x + blockIdx.x * blockDim.x;

    sh_A[0][threadIdx.x] = global_idx < N ? A[global_idx] : 0;
    sh_idxs[0][threadIdx.x] = (global_idx < N) ? old_idxs[global_idx] : 0;
    // 2. perform one bit radix
    bool input_idx = 0;
    #pragma unroll
    for(int r = 0;r < RADIX;r++) {
        one_bit_sort(sh_A[input_idx], sh_A[!input_idx], sh_idxs[input_idx], sh_idxs[!input_idx], sh_mem, iter*RADIX + r, op);
        input_idx = !input_idx;
    } 
    // 3. write it ^ write on global counter
    update_glob_buckets(sh_A[input_idx], local_counter, iter, N, global_idx, global_counter, op);
    if(global_idx < N) {
        res[global_idx] = sh_A[input_idx][threadIdx.x];
        new_idxs[global_idx] = sh_idxs[input_idx][threadIdx.x];
    }
}

// global_counter[radix][blockIdx.x]
template <typename OP>
__global__ void radix_sort_shuffle(uint32_t* A, uint32_t* res, uint32_t* old_idxs, uint32_t* new_idxs, uint32_t* global_counter, uint32_t* global_counter_sum, const uint32_t N, const uint32_t iter) {
    OP op;
    __shared__ int local_counter[BUCKET_SZ];
    if(threadIdx.x < BUCKET_SZ) {
        local_counter[threadIdx.x] = global_counter[GET_IDX(threadIdx.x, blockIdx.x, gridDim.x)];
    }
    __syncthreads();
    // sort locally
    if(threadIdx.x == 0) {
        for(int i = 1; i < BUCKET_SZ;i++) {
            local_counter[i] += local_counter[i - 1];
        }
    }
    __syncthreads();
    //
    int global_idx = threadIdx.x + blockIdx.x * blockDim.x;
    if(global_idx < N) {
        int radix = op.get_radix(A[global_idx], iter);
        int bucket_idx = radix == 0 ? threadIdx.x : threadIdx.x - local_counter[radix - 1];
        
        int counter_idx = GET_IDX(radix, blockIdx.x, gridDim.x) - 1;
        int sum = counter_idx >= 0 ? global_counter_sum[counter_idx] : 0;
        int new_idx = bucket_idx + sum;
        // printf("blcokIdx %i radix %i bucket_idx %i sum %i newidx %i\n",blockIdx.x, radix, bucket_idx, sum, new_idx);
        // printf("old: %i new: %i\n", global_idx, new_idx);
        res[new_idx] = A[global_idx];
        new_idxs[new_idx] = old_idxs[global_idx];
    }
}

int main() {
    // const int N = BLOCK_DIM;
    // int h_input[N] = {15, 3, 7, 9, 2, 1, 5, 6, 4, 8, 10, 0, 11, 13, 12, 14, 19, 17, 16, 18, 22, 20, 21, 23, 25, 24, 27, 26, 29, 28, 31, 30};
    // int h_output[N];
    const uint32_t N = 17;
    float h_input[N] = {12.5, 3.67, 3.4, 3.45,  -15, 8, 5, 10,  9, 6, 11, 13,  4,10,7,0, -5 };
    float h_output[N];
    
    uint32_t indexs[N], indexs_out[N];
    for(int i = 0; i < N;i++) indexs[i] = i;
    
    float *d_input_buff;
    uint32_t *d_input, *d_output, *d_idxs_in, *d_idxs_out;
    CUDA_CHECK(hipMalloc((void**)&d_input_buff, N * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_input, N * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_output, N * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_idxs_in, N * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_idxs_out, N * sizeof(int)));


    CUDA_CHECK(hipMemcpy(d_input_buff, h_input, N * sizeof(float), hipMemcpyHostToDevice));
    float_to_uint32<<<CEIL_DIVI(N, BLOCK_DIM), BLOCK_DIM>>>(d_input_buff, d_input, N);
    // CUDA_CHECK(hipMemcpy(d_input, h_input, N * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_idxs_in, indexs, N * sizeof(int), hipMemcpyHostToDevice));



    const int num_iters = 32/RADIX; // Enough for sorting up to 32-bit ints (since RADIX = 2, 16 * 2 = 32 bits)

    // 1. init global counter
    uint32_t* global_counter, *global_counter_sum;
    dim3 blocks( CEIL_DIV(N, BLOCK_DIM) );
    uint32_t counter_sz = blocks.x * BUCKET_SZ;
    dim3 blocks_counter( CEIL_DIV(counter_sz, 2*BLOCK_DIM) );
    // for global counter
    CUDA_CHECK( hipMalloc((void**)&global_counter, counter_sz * sizeof(int)) );
    CUDA_CHECK( hipMalloc((void**)&global_counter_sum, counter_sz * sizeof(int)) );//+1 to handle exclusive
    CUDA_CHECK( hipMemset(global_counter_sum, 0, sizeof(int)) );//+1 to handle exclusive
    // 2. init prefix sum on global counter
    uint32_t* block_counter, *blocks_finished;
    CUDA_CHECK( hipMalloc((void**)&block_counter, sizeof(int)) );
    CUDA_CHECK( hipMalloc((void**)&blocks_finished, sizeof(int)) );

    for (int iter = 0; iter < num_iters; ++iter) {
        CUDA_CHECK( hipMemset(block_counter, 0, sizeof(int)) );
        CUDA_CHECK( hipMemset(blocks_finished, 0, sizeof(int)) );
        
        radix_sort_local_kerenl<DescOp><<<blocks, BLOCK_DIM>>>(d_input, d_output, d_idxs_in, d_idxs_out, global_counter, N, iter);
        // prefix sum on global counter
        balanced_bank_kernel<<<blocks_counter, BLOCK_DIM>>>(global_counter, global_counter_sum, counter_sz, block_counter, blocks_finished);
        // update/global shuffling
        radix_sort_shuffle<DescOp><<<blocks, BLOCK_DIM>>>(d_output, d_input, d_idxs_out, d_idxs_in, global_counter, global_counter_sum, N, iter);
    }
    
    

    // CUDA_CHECK(hipMemcpy(h_output, d_input, N * sizeof(int), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(indexs_out, d_idxs_in, N * sizeof(int), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipDeviceSynchronize());
    for(int i = 0; i < N;i++) h_output[i] = h_input[ indexs_out[i] ];

    std::cout << "Sorted output:\n";
    for (int i = 0; i < N; ++i) std::cout << h_output[i] << " ";
    std::cout << std::endl;

    // Validate
    std::sort(h_input, h_input + N);
    std::reverse(h_input, h_input + N);
    for (int i = 0; i < N; ++i) {
        if (h_input[i] != h_output[i]) {
            std::cerr << "Test failed at index " << i << ": expected " << h_input[i] << ", got " << h_output[i] << std::endl;
            exit(EXIT_FAILURE);
        }
    }

    std::cout << "Test passed.\n";


    hipFree(d_input);
    hipFree(d_output);
    CUDA_CHECK(hipFree(d_idxs_in));
    CUDA_CHECK(hipFree(d_idxs_out));
    CUDA_CHECK(hipFree(block_counter));
    CUDA_CHECK(hipFree(blocks_finished));
    CUDA_CHECK(hipFree(global_counter));
    CUDA_CHECK(hipFree(global_counter_sum));
    
    return 0;
}